#include "hip/hip_runtime.h"
#include <vector>

#include "errors.hpp"

namespace happy {

__global__ void add_kernel(int* a, int* b, int* c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

std::vector<int> add_gpu(
    const std::vector<int>& v1, const std::vector<int>& v2) {
  ARGUMENT_CHECK(v1.size() == v2.size());

  int* d_a = nullptr;
  int* d_b = nullptr;
  int* d_c = nullptr;

  size_t n = v1.size();
  size_t alloc_size = sizeof(int) * n;
  std::vector<int> ret(n);

  hipMalloc(&d_a, alloc_size);
  hipMalloc(&d_b, alloc_size);
  hipMalloc(&d_c, alloc_size);

  hipMemcpy(d_a, v1.data(), alloc_size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, v2.data(), alloc_size, hipMemcpyHostToDevice);

  constexpr size_t kBlockSize = 256;
  size_t grid_size = 1 + (n - 1) / kBlockSize;
  add_kernel<<<grid_size, kBlockSize>>>(d_a, d_b, d_c, n);

  hipMemcpy(ret.data(), d_c, alloc_size, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return ret;
}

}  // namespace happy
